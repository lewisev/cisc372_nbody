#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include "vector.h"
#include "config.h"
#include "planets.h"
#include "compute.h"

// represents the objects in the system.  Global variables
vector3 *hVel, *d_hVel;
vector3 *hPos, *d_hPos;
double *mass, *d_mass;

vector3 **d_accels, *d_values;

//initHostMemory: Create storage for numObjects entities in our system
//Parameters: numObjects: number of objects to allocate
//Returns: None
//Side Effects: Allocates memory in the hVel, hPos, and mass global variables
//! does not change for parallelizing
void initHostMemory(int numObjects)
{
	hVel = (vector3 *)malloc(sizeof(vector3) * numObjects);
	hPos = (vector3 *)malloc(sizeof(vector3) * numObjects);
	mass = (double *)malloc(sizeof(double) * numObjects);
}

//freeHostMemory: Free storage allocated by a previous call to initHostMemory
//Parameters: None
//Returns: None
//Side Effects: Frees the memory allocated to global variables hVel, hPos, and mass.
//! does not change for parallelizing
void freeHostMemory()
{
	free(hVel);
	free(hPos);
	free(mass);
}

//planetFill: Fill the first NUMPLANETS+1 entries of the entity arrays with an estimation
//				of our solar system (Sun+NUMPLANETS)
//Parameters: None
//Returns: None
//Fills the first 8 entries of our system with an estimation of the sun plus our 8 planets.
//! does not change for parallelizing
void planetFill(){
	int i,j;
	double data[][7]={SUN,MERCURY,VENUS,EARTH,MARS,JUPITER,SATURN,URANUS,NEPTUNE};
	for (i=0;i<=NUMPLANETS;i++){
		for (j=0;j<3;j++){
			hPos[i][j]=data[i][j];
			hVel[i][j]=data[i][j+3];
		}
		mass[i]=data[i][6];
	}
}

//randomFill: FIll the rest of the objects in the system randomly starting at some entry in the list
//Parameters: 	start: The index of the first open entry in our system (after planetFill).
//				count: The number of random objects to put into our system
//Returns: None
//Side Effects: Fills count entries in our system starting at index start (0 based)
//! does not change for parallelizing
void randomFill(int start, int count)
{
	int i, j = start;
	for (i = start; i < start + count; i++)
	{
		for (j = 0; j < 3; j++)
		{
			hVel[i][j] = (double)rand() / RAND_MAX * MAX_DISTANCE * 2 - MAX_DISTANCE;
			hPos[i][j] = (double)rand() / RAND_MAX * MAX_VELOCITY * 2 - MAX_VELOCITY;
			mass[i] = (double)rand() / RAND_MAX * MAX_MASS;
		}
	}
}

//printSystem: Prints out the entire system to the supplied file
//Parameters: 	handle: A handle to an open file with write access to prnt the data to
//Returns: 		none
//Side Effects: Modifies the file handle by writing to it.
//! does not change for parallelizing
void printSystem(FILE* handle){
	int i,j;
	for (i=0;i<NUMENTITIES;i++){
		fprintf(handle,"pos=(");
		for (j=0;j<3;j++){
			fprintf(handle,"%lf,",hPos[i][j]);
		}
		printf("),v=(");
		for (j=0;j<3;j++){
			fprintf(handle,"%lf,",hVel[i][j]);
		}
		//fprintf(handle,"),m=%lf\n",mass[i]); //todo: uncomment this line to show mass
	}
}

int main(int argc, char **argv)
{
	clock_t t0=clock();
	int t_now;
	//srand(time(NULL));
	srand(1234);
	initHostMemory(NUMENTITIES);
	planetFill();
	randomFill(NUMPLANETS + 1, NUMASTEROIDS);
	//now we have a system.
	#ifdef DEBUG
	printSystem(stdout);
	#endif

	// Allocate memory
	hipMalloc((void**) &d_hVel, sizeof(vector3) * NUMENTITIES);
	hipMalloc((void**) &d_hPos, sizeof(vector3) * NUMENTITIES);
	hipMalloc((void**) &d_mass, sizeof(double) * NUMENTITIES);

	// create values and accels
	///Changed these to hipMalloc on host,dont need them done everytime in compute loop
	vector3* values = (vector3*) malloc(sizeof(vector3) * NUMENTITIES*NUMENTITIES);
	vector3** accels = (vector3**) malloc(sizeof(vector3*) * NUMENTITIES);

	//make an acceleration matrix which is NUMENTITIES squared in size;
	for (int i=0; i < NUMENTITIES; i++) {
		accels[i] =& values[i * NUMENTITIES];
	}

	hipMalloc((void**) &d_values, sizeof(vector3) * NUMENTITIES);
	hipMalloc((void**) &d_accels, sizeof(vector3*) * NUMENTITIES * NUMENTITIES);

	// Copy variables from host to device
	hipMemcpy(d_hVel, hVel, sizeof(vector3) * NUMENTITIES, hipMemcpyHostToDevice);
	hipMemcpy(d_hPos, hPos, sizeof(vector3) * NUMENTITIES, hipMemcpyHostToDevice);
	hipMemcpy(d_mass, mass, sizeof(double) * NUMENTITIES, hipMemcpyHostToDevice);

	hipMemcpy(d_values, values, sizeof(vector3) * NUMENTITIES*NUMENTITIES, hipMemcpyHostToDevice);
	hipMemcpy(d_accels, accels, sizeof(vector3) * NUMENTITIES, hipMemcpyHostToDevice);

	//* Call compute
	int count = 0;
	for (t_now=0;t_now<DURATION;t_now+=INTERVAL) {
		compute();
		count++;
	}

	// Copy variables from device to host
	hipMemcpy(hVel, d_hVel, sizeof(vector3) * NUMENTITIES, hipMemcpyDeviceToHost);
	hipMemcpy(hPos, d_hPos, sizeof(vector3) * NUMENTITIES, hipMemcpyDeviceToHost);

	//free all cuda memory
	hipFree(d_hVel);
	hipFree(d_hPos);
	hipFree(d_mass);
	hipFree(d_values);
	hipFree(d_accels);

	free(values);
	free(accels);

	clock_t t1=clock()-t0;
#ifdef DEBUG
	printSystem(stdout);

	//* debug
	printf("COUNTER:  %d\n", count);
#endif
	printf("This took a total time of %f seconds\n",(double)t1/CLOCKS_PER_SEC);

	freeHostMemory();
}
